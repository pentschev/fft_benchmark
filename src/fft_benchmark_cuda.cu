/*********************************************************************************
 * Copyright (c) 2015, Peter Andreas Entschev
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * * Redistributions of source code must retain the above copyright notice, this
 *   list of conditions and the following disclaimer.
 *
 * * Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * * Neither the name of fft_benchmark nor the names of its
 *   contributors may be used to endorse or promote products derived from
 *   this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *********************************************************************************/

#include "common.hpp"

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include <cstdlib>
#include <cstdio>
#include <iostream>

template< typename T >
void check(T result, char const *const func, const char *const file, int const line)
{
    if (result)
    {
        fprintf(stderr, "CUDA error at %s:%d code=%d \"%s\" \n",
                file, line, static_cast<unsigned int>(result), func);
        hipDeviceReset();
        exit(-1);
    }
}

#define CUDA_CHECK(val)           check ( (val), #val, __FILE__, __LINE__ )

static const int NSamplesMin = 128;
static const int NSamplesMax = 4096;
static const int Batches = 10;
static const int Rank = 2;
static const int Iterations = 10;

int main()
{
#if __cplusplus > 199711L
        std::chrono::high_resolution_clock::time_point tStart, tEnd;
#else
        std::clock_t tStart, tEnd;
#endif

    for (int n = NSamplesMin; n <= NSamplesMax; n <<= 1) {
        hipfftHandle plan_c2c;
        hipfftComplex *d_complex, *d_c2c_out;

        size_t in_sz = n;
        for (int i = 1; i < Rank; i++)
            in_sz *= n;

        int dims[] = {n, n, n};

        size_t bufferSize = in_sz * Batches * sizeof(hipfftComplex);

        // No data is copied to buffers as FFT performance is not data
        // dependent, but only size dependent
        CUDA_CHECK(hipMalloc((void**)&d_complex, bufferSize));
        CUDA_CHECK(hipMalloc((void**)&d_c2c_out, bufferSize));

        // Allocate cuFFT plan
        CUDA_CHECK(hipfftPlanMany(&plan_c2c, Rank, dims, NULL, 0, 0,
                                 NULL, 0, 0, HIPFFT_C2C, Batches));

        std::cout << "Number of dimensions: " << Rank << std::endl;
        std::cout << "Matrix dimensions: " << n << "x" << n << std::endl;
        std::cout << "Batch size: " << Batches << std::endl << std::endl;

        tStart = getTime();
        for (int i = 0; i < Iterations; i++) {
            // In-place plan
            CUDA_CHECK(hipfftExecC2C(plan_c2c, d_c2c_out, d_c2c_out, HIPFFT_FORWARD));
        }
        CUDA_CHECK(hipDeviceSynchronize());
        tEnd = getTime();
        std::cout << "In-place C2C FFT time for " << Iterations << " runs: " << getTimeCount(tEnd, tStart) << " ms" << std::endl;

        tStart = getTime();
        for (int i = 0; i < Iterations; i++) {
            // Out-of-place plan
            CUDA_CHECK(hipfftExecC2C(plan_c2c, d_complex, d_c2c_out, HIPFFT_FORWARD));
        }
        CUDA_CHECK(hipDeviceSynchronize());
        tEnd = getTime();
        std::cout << "Out-of-place C2C FFT time for " << Iterations << " runs: " << getTimeCount(tEnd, tStart) << " ms" << std::endl;

        tStart = getTime();
        for (int i = 0; i < Iterations; i++) {
            CUDA_CHECK(hipMemcpy(d_complex, d_c2c_out, bufferSize, hipMemcpyDeviceToDevice));
            // Out-of-place plan
            CUDA_CHECK(hipfftExecC2C(plan_c2c, d_complex, d_c2c_out, HIPFFT_FORWARD));
        }
        CUDA_CHECK(hipDeviceSynchronize());
        tEnd = getTime();
        std::cout << "Buffer Copy + Out-of-place C2C FFT time for " << Iterations << " runs: " << getTimeCount(tEnd, tStart) << " ms" << std::endl << std::endl;

        // Destroy plan
        CUDA_CHECK(hipfftDestroy(plan_c2c));

        // Free CUDA buffers
        CUDA_CHECK(hipFree(d_complex));
        CUDA_CHECK(hipFree(d_c2c_out));
    }
}
